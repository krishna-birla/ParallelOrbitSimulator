#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <stdbool.h>
#include <time.h>
#include <limits.h>
#include <hip/hip_runtime.h>


#define ORBITSPERDAY 20.0
#define SIMULATIONTIMEDAYS 1
#define STEPTIME 50.0
#define SECONDSPERDAY 86400.0
#define PI 3.141592653589793
#define RADIUSOFORBIT 7000.0
#define INCLINATION 30.0

double batterycharge = 100.00;
int num_images = 0;
double avg_charge = 0.0;
int num_images_comp = 0;
int num_images_trans = 0;
int num_beacon_trans = 0;
double sun_time = 0.0;
int gps_access = 0;
int batteryfails = 0;
int adcsfails = 0;
double average_comp_ratio = 0.0;

__global__ void masterkernel(int* imgsizecuda,double* pos, int* suncuda, uint16_t* imgcuda, uint8_t* compimgcuda, double* batterychargecuda,\
                                int* batteryfailscuda, double* avg_chargecuda, int* adcsfailscuda, int* gps_accesscuda,\
                                int* num_imagescuda, int* num_images_transcuda, int* num_images_compcuda,\
                                double* average_comp_ratiocuda, int* num_beacon_transcuda,\
                                double* sun_timecuda, double SECONDSPERORBIT, double inc, double sec)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    switch(bid)
    {
        case 0:{
            switch(tid)
            {
                case 0:{
                    if(*batterychargecuda < 30.00)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    double theta = sec * (360.0 / SECONDSPERORBIT);
                    double phi = inc;
                    double range = sqrt(pow(pos[0], 2) + pow(pos[1], 2) + pow(pos[2], 2));
                    double xaxis = range * cos(theta * (PI / 180.0));
                    double yaxis = range * sin(theta * (PI / 180.0)) * cos(phi * (PI / 180.0));
                    double zaxis = range * sin(theta * (PI / 180.0)) * sin(phi * (PI / 180.0));
                    pos[0] = xaxis;
                    pos[1] = yaxis;
                    pos[2] = zaxis;
                    (gps_accesscuda)++;
                    *batterychargecuda -= 12.0;
                    break;
                }
                case 1:{
                    if(*batterychargecuda < 10.0)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    if(sec > (SECONDSPERORBIT / 2))
                    {
                        return;
                    }
                    int ind1 = 0;
                    int ind2 = 3;
                    int ind3 = 4;
                    suncuda[ind1] = suncuda[ind2] = suncuda[ind3] = 1;
                    *sun_timecuda += 0.1 * ((suncuda[0] ? 1 : 0) + (suncuda[5] ? 1 : 0)) + 0.2 * ((suncuda[1] ? 1 : 0) + (suncuda[2] ? 1 : 0) + (suncuda[3] ? 1 : 0) + (suncuda[4] ? 1 : 0));
                    *batterychargecuda -= 6.0;
                    break;
                }
                case 2:{
                    if(*batterychargecuda < 40.0)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    int i = 0;
                    for(i = 0;i < 512;i++)
                    {
                        int j = 0;
                        for(j = 0;j < 640;j++)
                        {
                            imgcuda[i * 640 + j] = 5524;
                        }
                    }
                    (*num_imagescuda)++;
                    *batterychargecuda -= 25.0;
                    break;
                }
                case 3:{
                    if(*batterychargecuda < 6.0)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    int i = 0, mainindex = 0;
                    for(i = 0;i < (512 * 640);i += 2)
                    {
                        uint8_t first = (imgcuda[i] & 0x0000FFFF);
                        uint8_t second = (((uint16_t)(imgcuda[i] >> 8)) & 0x0000FFFF);
                        compimgcuda[mainindex++] = first + second;
                    }
                    *batterychargecuda -= 5.0;
                    (*num_images_compcuda)++;
                    *imgsizecuda = mainindex;
                    *average_comp_ratiocuda += (*imgsizecuda) > 0.0 ? ((512.0 * 640.0 * 2) / ((double)(*imgsizecuda))) : 0;
                    break;
                }
            }
            break;
        }
        case 1:{
            switch(tid)
            {
                case 0:{
                    if(*batterychargecuda < 25.0)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    int i;
                    for(i = 0;i < 3;i++)
                    {
                        int j;
                        for(j = 0;j < *imgsizecuda;j++)
                        {
                            compimgcuda[j] = 0;
                        }
                    }
                    *batterychargecuda -= 15.0;
                    (*num_images_compcuda)++;
                    break;
                }
                case 1:{
                    if(*batterychargecuda < 20.0)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    int i;
                    uint8_t becaon[38];
                    union DBL
                    {
                        double d;
                        char c[sizeof(double)];
                    }dbl;
                    for(i = 0;i < 60;i++)
                    {
                        int mainindex = 0;
                        int j;
                        for(j = 0;j < 3;j++)
                        {
                            dbl.d = pos[j];
                            int k = 0;
                            for(k = 0;k < sizeof(double);k++)
                            {
                                becaon[mainindex++] = dbl.c[k];
                            }
                        }
                        for(j = 0;j < 6;j++)
                        {
                            becaon[mainindex++] = (suncuda[j] ? 1 : 0);
                        }
                        dbl.d = *batterychargecuda;
                        for(j = 0;j < sizeof(double);j++)
                        {
                            becaon[mainindex++] = dbl.c[j];
                        }
                        for(j = 0;j < 38;j++)
                        {
                            becaon[j] = 0;
                        }
                    }
                    (*num_beacon_transcuda)++;
                    *batterychargecuda -= 10.0;
                    break;
                }
                case 2:{
                    if(*batterychargecuda < 6.0)
                    {
                        (*batteryfailscuda)++;
                        return;
                    }
                    double range = sqrt(pow(pos[0], 2) + pow(pos[1], 2) + pow(pos[2], 2));
                    double theta = acos(pos[0] / range) * (180.0 / PI);
                    double phi = acos(pos[1] / (range * sin(theta * (PI / 180.0)))) * (180.0 / PI);
                    phi = (phi + (asin(pos[2] / (range * sin(theta * (PI / 180.0)))) * (180.0 / PI))) / 2;
                    if(((int)(inc)) != ((int)(phi)))
                    {
                        (*adcsfailscuda)++;
                    }
                    *batterychargecuda -= 5.0;
                    break;
                }
                case 3:{
                    if(sec <= (SECONDSPERORBIT / 2))
                    {
                        *batterychargecuda = min(100.0, *batterychargecuda + 60.0);
                        *avg_chargecuda += *batterychargecuda;
                    }
                    break;
                }
            }
            break;
        }
    }
}

int main()
{
    printf("****Orbit Simulator****\n\n");
    printf("Geocentric circular orbit\n");
    printf("Radius of orbit: %lfkm\n", RADIUSOFORBIT);
    printf("Height of orbit: %lfkm\n", RADIUSOFORBIT - 6400.0);
    printf("Inclination of orbit: %lfdeg\n", INCLINATION);
    printf("Number of orbits per day: %lf\n", ORBITSPERDAY);
    printf("Temporal length of each orbit: %lfsec\n", SECONDSPERDAY / ORBITSPERDAY);
    printf("Tangential orbital velocity: %lfkm/sec\n\n", (2 * PI * RADIUSOFORBIT) / (SECONDSPERDAY / ORBITSPERDAY));
    double Position[3] = {RADIUSOFORBIT, 0.0, 0.0};
    uint16_t image[512][640] = {0};
    uint8_t compressedimage[512 * 640] = {0};
    int SunSensorVal[6] = {0};
    int days = 0;

    int *suncuda, *imgsizecuda, *num_imagescuda, *num_images_compcuda, *num_images_transcuda, *num_beacon_transcuda, *batteryfailscuda, *adcsfailscuda, *gps_accesscuda;
    uint8_t* compimgcuda;
    uint16_t* imgcuda;
    int compressedsize = 0.0;
    double *avg_chargecuda, *sun_timecuda, *batterychargecuda, *poscuda, *average_comp_ratiocuda;

    hipMalloc(&poscuda, sizeof(double) * 3);
    hipMalloc(&suncuda, sizeof(int) * 6);
    hipMalloc(&imgcuda, sizeof(uint16_t) * 512 * 640);
    hipMalloc(&compimgcuda, sizeof(uint8_t) * 512 * 640);
    hipMalloc(&num_images_compcuda, sizeof(int));
    hipMalloc(&num_imagescuda, sizeof(int));
    hipMalloc(&num_images_transcuda, sizeof(int));
    hipMalloc(&gps_accesscuda, sizeof(int));
    hipMalloc(&num_beacon_transcuda, sizeof(int));
    hipMalloc(&batteryfailscuda, sizeof(int));
    hipMalloc(&imgsizecuda, sizeof(int));
    hipMalloc(&adcsfailscuda, sizeof(int));
    hipMalloc(&avg_chargecuda, sizeof(double));
    hipMalloc(&average_comp_ratiocuda, sizeof(double));
    hipMalloc(&batterychargecuda, sizeof(double));
    hipMalloc(&sun_timecuda, sizeof(double));

    hipMemcpy(poscuda, Position, sizeof(double) * 3, hipMemcpyHostToDevice);
    hipMemcpy(avg_chargecuda, &avg_charge, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(average_comp_ratiocuda, &average_comp_ratio, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(sun_timecuda, &sun_time, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(suncuda, SunSensorVal, sizeof(int) * 6, hipMemcpyHostToDevice);
    hipMemcpy(batterychargecuda, &batterycharge, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(batteryfailscuda, &batteryfails, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(imgsizecuda, &compressedsize, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(adcsfailscuda, &adcsfails, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(imgcuda, image, sizeof(uint16_t) * 512 * 640, hipMemcpyHostToDevice);
    hipMemcpy(compimgcuda, compressedimage, sizeof(uint8_t) * 512 * 640, hipMemcpyHostToDevice);
    hipMemcpy(gps_accesscuda, &gps_access, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(num_imagescuda, &num_images, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(num_beacon_transcuda, &num_beacon_trans, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(num_images_transcuda, &num_images_trans, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(num_images_compcuda, &num_images_comp, sizeof(int), hipMemcpyHostToDevice);

    clock_t begin = clock();

    for(days = 1;days <= SIMULATIONTIMEDAYS;days++)
    {
        printf("%s%d\n", "Day: ", days);
        double orbits = 0;
        for(orbits = 0.0;orbits < ORBITSPERDAY;orbits++)
        {
            printf("%s%lf\n", "Orbit: ", orbits);
            double seconds = 0.0;
            double SECONDSPERORBIT = SECONDSPERDAY / ORBITSPERDAY;
            for(seconds = 0.0;seconds <= SECONDSPERORBIT;seconds += STEPTIME)
            {
                masterkernel<<<2, 4>>>(imgsizecuda, poscuda, suncuda, imgcuda,\
                                        compimgcuda, batterychargecuda, batteryfailscuda, avg_chargecuda,\
                                        adcsfailscuda, gps_accesscuda, num_imagescuda, num_images_transcuda,\
                                        num_images_compcuda, average_comp_ratiocuda, num_beacon_transcuda,\
                                        sun_timecuda, SECONDSPERORBIT, INCLINATION, seconds);
            }
        }
    }

    clock_t end = clock();

    hipMemcpy(Position, poscuda, sizeof(double) * 3, hipMemcpyDeviceToHost);
    hipMemcpy(&avg_charge, avg_chargecuda, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&average_comp_ratio, average_comp_ratiocuda, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&sun_time, sun_timecuda, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(SunSensorVal, suncuda, sizeof(int) * 6, hipMemcpyDeviceToHost);
    hipMemcpy(&batterycharge, batterychargecuda, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&batteryfails, batteryfailscuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&adcsfails, adcsfailscuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&compressedsize, imgsizecuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(image, imgcuda, sizeof(uint16_t) * 512 * 640, hipMemcpyDeviceToHost);
    hipMemcpy(compressedimage, compimgcuda, sizeof(uint8_t) * 512 * 640, hipMemcpyDeviceToHost);
    hipMemcpy(&gps_access, gps_accesscuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&num_images, num_imagescuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&num_beacon_trans, num_beacon_transcuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&num_images_trans, num_images_transcuda, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&num_images_comp, num_images_compcuda, sizeof(int), hipMemcpyDeviceToHost);

    printf("\nRelative sun time: %lf\n", sun_time);
    printf("Number of GPS Access: %d\n", gps_access);
    printf("Number of images clicked: %d\n", num_images);
    printf("Number of images compressed: %d\n", num_images_comp);
    printf("Number of images transmitted: %d\n", num_images_trans);
    printf("Average compression ratio: %lf\n", average_comp_ratio / ((double)(num_images_comp)));
    printf("Number of beacon transmissions: %d\n", num_beacon_trans);
    printf("Number of ADCS failures: %d\n", adcsfails);
    printf("Average battery charge: %lf\n", avg_charge / ((double)(SIMULATIONTIMEDAYS * SECONDSPERDAY)));
    printf("Number of battery failures: %d\n", batteryfails);
    printf("Run time: %lf\n", ((double)(end - begin)) / CLOCKS_PER_SEC);
}
